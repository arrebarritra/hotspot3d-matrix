#include "hip/hip_runtime.h"
#include <hipsparse.h>
#include <hipblas.h>

#define gpuCheck(stmt)                                               \
  do {                                                               \
      hipError_t err = stmt;                                        \
      if (err != hipSuccess) {                                      \
          printf("%s. Failed to run stmt %s\n",	hipGetErrorName(err), #stmt);           \
          break;                                                     \
      }                                                              \
  } while (0)

// Macro to check the cuBLAS status
#define cublasCheck(stmt)                                            \
  do {                                                               \
      hipblasStatus_t err = stmt;                                     \
      if (err != HIPBLAS_STATUS_SUCCESS) {                            \
          printf("ERROR. Failed to run cuBLAS stmt %s\n", #stmt);    \
          break;                                                     \
      }                                                              \
  } while (0)

// Macro to check the cuSPARSE status
#define cusparseCheck(stmt)                                          \
  do {                                                               \
      hipsparseStatus_t err = stmt;                                   \
      if (err != HIPSPARSE_STATUS_SUCCESS) {                          \
          printf("ERROR. Failed to run cuSPARSE stmt %s\n", #stmt);  \
          break;                                                     \
      }                                                              \
  } while (0)

long long get_time() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (tv.tv_sec * 1000000) + tv.tv_usec;
}

// -----------
// GLOBAL VARS
// -----------

const float zero = 0.0f;
const float one = 1.0f;

// Differentation matrices
float *dX, *dY, *dZ;
// Note: dX and dY will have same RowPtr and ColIndx
int *dXYRowPtr, *dZRowPtr;
int *dXYColIndx, *dZColIndx;
// Device data (use ping pong strategy for input/output temp)
float *t_d[2], *p_d;
// cuSPARSE handle
hipsparseHandle_t cusparseHandle;
// cuSPARSE diff matrix descriptions
hipsparseSpMatDescr_t dXDescr, dYDescr, dZDescr;
// cuSPARSE data matrix descriptions (note ping pong strategy is used for T descr)
hipsparseDnMatDescr_t tXYDescr[2], tXYZDescr[2],
                     pXYDescr, pXYZDescr;
// cuSPARSE calculation buffers
size_t bufferSizeX, bufferSizeY, bufferSizeZ;
float *bufferX, *bufferY, *bufferZ;
// cuBLAS handle
hipblasHandle_t cublasHandle;

// ------------
// CALCULATIONS
// ------------

// Differentiation matrix, stencil is adapted for orientation
void diffMatrixInit(float* A, int* ArowPtr, int* AcolIndx,
    int rows, float stencil[3]) {
  // Variable holding the position to insert a new element
  size_t ptr = 0;
  ArowPtr[0] = ptr;

  // Configure first row (2 elements due to boundary)
  A[ptr] = stencil[0] + stencil[1];
  AcolIndx[ptr++] = 0;
  A[ptr] = stencil[2];
  AcolIndx[ptr++] = 1;
  ArowPtr[1] = ptr;

  // Fill middle of the matrix
  for (int i = 1; i < (rows - 1); ++i) {
    for (int k = 0; k < 3; ++k) {
      A[ptr] = stencil[k];
      AcolIndx[ptr++] = i + k - 1;
    }
    ArowPtr[i + 1] = ptr;
  }

  // Configure last row (2 elements due to boundary)
  A[ptr] = stencil[0];
  AcolIndx[ptr++] = rows - 2;
  A[ptr] = stencil[1] + stencil[2];
  AcolIndx[ptr++] = rows - 1;
  ArowPtr[rows] = ptr;
}

void cusparseDiffMatConfig(int nx, int ny, int nz, int nzv_xy, int nzv_z) {
    cusparseCheck(hipsparseCreateCsr(&dXDescr, nx, nx, nzv_xy,
            dXYRowPtr, dXYColIndx, dX,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
            HIP_R_32F));
    cusparseCheck(hipsparseCreateCsr(&dYDescr, ny, ny, nzv_xy,
            dXYRowPtr, dXYColIndx, dY,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
            HIP_R_32F));
    cusparseCheck(hipsparseCreateCsr(&dZDescr, nz, nz, nzv_z,
            dZRowPtr, dZColIndx, dZ,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
            HIP_R_32F));

    // dummy strided batches
    cusparseCheck(hipsparseCsrSetStridedBatch(dXDescr, nz, 0, 0));
    cusparseCheck(hipsparseCsrSetStridedBatch(dYDescr, nz, 0, 0));
}

void cusparseDataMatConfig(int nx, int ny, int nz) {
    cusparseCheck(hipsparseCreateDnMat(&tXYDescr[0], ny, nx, nx,
            t_d[0], HIP_R_32F, HIPSPARSE_ORDER_ROW));
    cusparseCheck(hipsparseCreateDnMat(&tXYDescr[1], ny, nx, nx,
            t_d[1], HIP_R_32F, HIPSPARSE_ORDER_ROW));
    cusparseCheck(hipsparseCreateDnMat(&pXYDescr, ny, nx, nx,
            p_d, HIP_R_32F, HIPSPARSE_ORDER_ROW));
    // Flattened version of data for operation by the diff Z matrix
    cusparseCheck(hipsparseCreateDnMat(&tXYZDescr[0], nz, nx * ny, nx * ny,
            t_d[0], HIP_R_32F, HIPSPARSE_ORDER_ROW)); 
    cusparseCheck(hipsparseCreateDnMat(&tXYZDescr[1], nz, nx * ny, nx * ny,
            t_d[1], HIP_R_32F, HIPSPARSE_ORDER_ROW));
    cusparseCheck(hipsparseCreateDnMat(&pXYZDescr, nz, nx * ny, nx * ny,
            p_d, HIP_R_32F, HIPSPARSE_ORDER_ROW));

    // Use strided batches to create XY matrix for all layers
    cusparseCheck(hipsparseDnMatSetStridedBatch(tXYDescr[0], nz, nx * ny));
    cusparseCheck(hipsparseDnMatSetStridedBatch(tXYDescr[1], nz, nx * ny));
}

// Calculate and allocate calculation buffer
void cusparseCalcBufferAlloc() {
    // Note: calculation buffers will be reused as ping pong buffers
    // Note: Buffer size symmetrical between t___[0] and t___[1],
    //       hence only one calc performed 
    cusparseCheck(hipsparseSpMM_bufferSize(cusparseHandle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    HIPSPARSE_OPERATION_TRANSPOSE,
                    &one, dXDescr, tXYDescr[0],
                    &one, tXYDescr[1],
                    HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG2,
                    &bufferSizeX)
                );
    cusparseCheck(hipsparseSpMM_bufferSize(cusparseHandle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &one, dYDescr, tXYDescr[0],
                    &one, tXYDescr[1],
                    HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG2,
                    &bufferSizeY)
                );
    cusparseCheck(hipsparseSpMM_bufferSize(cusparseHandle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &one, dZDescr, tXYZDescr[0],
                    &one, tXYZDescr[1],
                    HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG2,
                    &bufferSizeZ)
                );
    
    gpuCheck(hipMallocManaged(&bufferX, bufferSizeX));
    gpuCheck(hipMallocManaged(&bufferY, bufferSizeY));
    gpuCheck(hipMallocManaged(&bufferZ, bufferSizeZ));
}

// Perform cuSPARSE part of calculation
void cusparseCalc(int in) {
    int out = !in;

    cusparseCheck(hipsparseSpMM(cusparseHandle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    HIPSPARSE_OPERATION_TRANSPOSE,
                    &one, dXDescr, tXYDescr[in],
                    &one, tXYDescr[out],
                    HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG2,
                    bufferX)
                );
    hipDeviceSynchronize();

    cusparseCheck(hipsparseSpMM(cusparseHandle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &one, dYDescr, tXYDescr[in],
                    &one, tXYDescr[out],
                    HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG2,
                    bufferY)
                );
    hipDeviceSynchronize();

    cusparseCheck(hipsparseSpMM(cusparseHandle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    &one, dZDescr, tXYZDescr[in],
                    &one, tXYZDescr[out],
                    HIP_R_32F, HIPSPARSE_SPMM_CSR_ALG2,
                    bufferZ)
                );
    hipDeviceSynchronize();
}

// Perform cuBLAS part of calculation
void cublasCalc(int out, int nx, int ny, int nz, float sdc) {
    cublasCheck(hipblasSaxpy(cublasHandle, nx * ny * nz,
                    &sdc, p_d, 1, t_d[out], 1));
    hipDeviceSynchronize();       
}

__global__ void addAmbTemp(float* tOut, float ct, float ambTemp) {
    int blockId = blockIdx.x
            + blockIdx.y * gridDim.x
            + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * blockDim.x + threadIdx.x;

    tOut[threadId] += ct * ambTemp;
}

void hotspot_opt1(float *p, float *tIn, float *tOut,
        int nx, int ny, int nz,
        float Cap, 
        float Rx, float Ry, float Rz, 
        float dt, int numiter) 
{
    long long start_setup = get_time();

    float ce, cw, cn, cs, ct, cb, cc;
    float stepDivCap = dt / Cap;
    ce = cw =stepDivCap/ Rx;
    cn = cs =stepDivCap/ Ry;
    ct = cb =stepDivCap/ Rz;

    cc = 1.0 - (2.0*ce + 2.0*cn + 3.0*ct);

    // Copy temp and power data to device
    size_t s = sizeof(float) * nx * ny * nz;  
    gpuCheck(hipMalloc(&p_d,s));
    gpuCheck(hipMalloc(&t_d[0],s));
    gpuCheck(hipMalloc(&t_d[1],s));
    gpuCheck(hipMemcpy(t_d[0], tIn, s, hipMemcpyHostToDevice));
    gpuCheck(hipMemcpy(p_d, p, s, hipMemcpyHostToDevice));

    // non-zero values for respective matrices
    int nzv_xy = 3 * (nx - 2) + 4;
    int nzv_z = 3 * (nz - 2) + 4;

    gpuCheck(hipMallocManaged(&dX, sizeof(float) * nzv_xy));
    gpuCheck(hipMallocManaged(&dY, sizeof(float) * nzv_xy));
    gpuCheck(hipMallocManaged(&dZ, sizeof(float) * nzv_z));
    gpuCheck(hipMallocManaged(&dXYRowPtr, sizeof(int) * (nx + 1)));
    gpuCheck(hipMallocManaged(&dZRowPtr, sizeof(int) * (nz + 1)));
    gpuCheck(hipMallocManaged(&dXYColIndx, sizeof(int) * nzv_xy));
    gpuCheck(hipMallocManaged(&dZColIndx, sizeof(int) * nzv_z));
    
    // Create CSR diff matrices
    // TODO: optimise by only adding cc component once
    float stencilX[3] = {cw, cc, ce};
    float stencilY[3] = {cn, 0.0f, cs};
    float stencilZ[3] = {cb, 0.0f, ct};
    diffMatrixInit(dX, dXYRowPtr, dXYColIndx, nx, stencilX);
    diffMatrixInit(dY, dXYRowPtr, dXYColIndx, ny, stencilY);
    diffMatrixInit(dZ, dZRowPtr, dZColIndx, nz, stencilZ);

    // Init cuSPARSE
    cusparseCheck(hipsparseCreate(&cusparseHandle));

    // Set up cuSPARSE matrices and calc buffers
    cusparseDiffMatConfig(nx, ny, nz, nzv_xy, nzv_z);
    cusparseDataMatConfig(nx, ny, nz);
    cusparseCalcBufferAlloc();

    // Init cuBLAS
    cublasCheck(hipblasCreate(&cublasHandle));
    cublasCheck(hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST));

    long long stop_setup = get_time();
    float time_setup = (float)((stop_setup - start_setup)/(1000.0 * 1000.0));
    printf("Time for setup: %.3f (s)\n",time_setup);

    long long start = get_time();
    int in = 0;
    dim3 blockDim(8, 8, 8);
    dim3 gridDim((nx + 8) / 8, (ny + 8) / 8, (nz + 8) / 8);
    float amb_temp = 80.0f;
    for (int i = 0; i < numiter; ++i) {
        cusparseCalc(in);
        cublasCalc(!in, nx, ny, nz, stepDivCap);
        addAmbTemp<<<gridDim, blockDim>>>(t_d[!in], ct, amb_temp);
        hipDeviceSynchronize();
        // Swap ping-pong buffers
        in = !in;
    }
    int out = in;
    
    long long stop = get_time();
    float time = (float)((stop - start)/(1000.0 * 1000.0));
    printf("Time: %.3f (s)\n",time);    
    hipMemcpy(tOut, t_d[out], s, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(p_d);
    hipFree(t_d[0]);
    hipFree(t_d[1]);
    return;
}

